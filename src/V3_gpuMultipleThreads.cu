#include "hip/hip_runtime.h"
/**************************************************************
* NOTE: This code respects the initial interface, so that it
* succesfully passes the online grader. The rest versions
* optimize for speed and space by using floats and 8-bit ints
* 
**************************************************************/ 
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include "../inc/ising.h"


#define diff 1e-6f
#define WINDOW_SIZE 5
#define MIN_MARGIN 5

// Debugging function that prints the first nXn elements of a sizeXsize array
void print_nn_array(int *x, int n, int size){
    for(int i=0; i<n; ++i){
        for(int j=0; j<n; ++j){
            printf("%d ", x[(i + 20)*size + j + 20]);
        }
        printf("\n");
    }
    printf("\n");
}

// CUDA Kernel
__global__ void computeMoment(int *readArr, int *writeArr, double *weightArr, int n){
    // The dimensions are hardcoded here to simplify extra syntax
    // cuda uses for dynamic shared memory allocation
    __shared__ int readArr_shared[32][32];
    __shared__ double weightArr_shared[5][5];

    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if(threadIdx.x<5 && threadIdx.y < 5){
        weightArr_shared[threadIdx.x][threadIdx.y] = weightArr[threadIdx.x*WINDOW_SIZE + threadIdx.y];
    }
    __syncthreads();

    // Only values within the below borders will be used but the __syncthreads()
    // function has to be called outside if statements so we load everything here
    readArr_shared[threadIdx.x][threadIdx.y] = readArr[row*n + col];
    __syncthreads();

    // If coordinates are between boundaries
    // update the write array accordingly
    if(row < n && col < n){
        float influence = 0.0f;
        for (int i=-2; i<3; i++)
        {
            for (int j=-2; j<3; j++)
            {
                //add extra n so that modulo behaves like mathematics modulo
                //that is return only positive values
                if(threadIdx.x >= MIN_MARGIN && threadIdx.y >= MIN_MARGIN && 
                    threadIdx.x <= 31-MIN_MARGIN && threadIdx.y <= 31-MIN_MARGIN){
                    int y = threadIdx.x + i;
                    int x = threadIdx.y + j;
                    influence += weightArr_shared[i+2][j+2]*readArr_shared[y][x];
                }else{
                    int y = (row+i+n)%n;
                    int x = (col+j+n)%n;
                    influence += weightArr_shared[i+2][j+2]*readArr[y*n + x];
                }
            }
        }

        if(threadIdx.x >= MIN_MARGIN && threadIdx.y >= MIN_MARGIN && 
            threadIdx.x <= 31-MIN_MARGIN && threadIdx.y <= 31-MIN_MARGIN){
            writeArr[row*n + col] = readArr_shared[threadIdx.x][threadIdx.y];
            if 		(influence<-diff)	writeArr[row*n + col] = -1;
            else if (influence>diff)	writeArr[row*n + col] = 1;
        }else {
            writeArr[row*n + col] = readArr[row*n + col];
            if 		(influence<-diff)	writeArr[row*n + col] = -1;
            else if (influence>diff)	writeArr[row*n + col] = 1;
        }
    }
    __syncthreads();

}

void ising(int *G, double *w, int k, int n)
{
    // Allocate memory for the 3 arrays with hipMallocManaged()
    // because they will be used inside the kernel
    // The return err values are for debugging only
    int *readArr, *writeArr;
    hipError_t err1 = hipMallocManaged(&readArr, n*n*sizeof(int));
    hipError_t err2 = hipMallocManaged(&writeArr,n*n*sizeof(int));
    double *weightArr_d;
    hipError_t er3 = hipMallocManaged(&weightArr_d, 5*5*sizeof(double));

    // Copy the contents of input arrays inside 
    // the ones we will use inside kernel
    memcpy(readArr, G, n*n*sizeof(int));
    memcpy(weightArr_d, w, 5*5*sizeof(double));

    for (int i=1; i<=k; i++)
    {
        // Create blocks of size 32x32 threads per block
        // The number of blocks will adjust to fit the input n
        dim3 dimBlock(32, 32);
        int gridSz = (n + 32)/ 32;
        dim3 dimGrid(gridSz, gridSz);

        // Run the kernel in GPU
        computeMoment<<<dimGrid, dimBlock>>> (readArr, writeArr, weightArr_d, n);

        // Uncomment below to check for launch errors
        //printf("%s\n", hipGetErrorString(hipGetLastError()));

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        // Swap read and write arrays
        int *temp = readArr;
        readArr = writeArr;
        writeArr = temp;
    }

    //The final result now is in readArr. Copy the contents
    // in array G
    memcpy(G, readArr, n*n*sizeof(int));

    hipFree( readArr     );
    hipFree( writeArr 	  );
    hipFree( weightArr_d );
}

int main()
{
    FILE* fin = fopen("../test/conf-init.bin","rb");
    FILE* fout = fopen("../test/conf-11-3.ans","wb");

    int n=517, k=11;

    double weights[5][5] = { {0.004f,0.016f,0.026f,0.016f,0.004f},
                            {0.016f,0.071f,0.117f,0.071f,0.016f},
                            {0.026f,0.117f,0.000f,0.117f,0.026f},
                            {0.016f,0.071f,0.117f,0.071f,0.016f},
                            {0.004f,0.016f,0.026f,0.016f,0.004f}};


    int *latticeArr = (int *) malloc(n*n*sizeof(int));

    //read from binary
    for (int row=0; row<n; row++)
    {
        for (int col=0; col<n; col++)
        {
            int spin;
            fread(&spin, sizeof(int), 1, fin);
            latticeArr[row*n + col] = spin;
        }
    }

    ising(latticeArr, (double*)weights, k, n);

    //write to binary
    for (int row=0; row<n; row++)
    {
        for (int col=0; col<n; col++)
        {
            int spin = latticeArr[row*n + col];
            fwrite(&spin, sizeof(int), 1, fout);
        }
    }

    free( latticeArr );
    return 0;
}